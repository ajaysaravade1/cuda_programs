
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<time.h>
__global__ void add_gpu(int *a, int *b,int *c)
{
 *c = *a + *b;
}

int main()
{
    int *h_a;
    int *h_b;
    int *h_c;

    clock_t begin,end;
    double time_spent;

    int *d_a,*d_b,*d_c;

    h_a = (int *)malloc(sizeof(int));
    h_b = (int *)malloc(sizeof(int));

    hipMalloc((void **)&d_a,sizeof(int));
    hipMalloc((void **)&d_b,sizeof(int));

    *h_a = 20;
    *h_b = 30;
    begin = clock();

    hipMemcpy(d_a,h_a,sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(d_a,h_a,sizeof(int),hipMemcpyHostToDevice);

    add_gpu<<<1,1>>>(d_a,d_b,d_c);
    hipMemcpy(h_c,d_c,sizeof(int),hipMemcpyDeviceToHost);       
    end = clock();

    time_spent = (double)(end-begin)/CLOCKS_PER_SEC;

    printf("\nTime spent simple Addition %f seconds",time_spent);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    free(h_a);
    free(h_b);
    free(h_c);
    
    
    return 0;
}