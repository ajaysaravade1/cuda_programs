
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<time.h>
int n;

__global__ void vector_add_gpu(int *a,int *b,int *c)
{
    int index  = (blockIdx.x * blockDim.x) + threadIdx.x;
    c[index] = a[index] +b[index];

}
void vector_add_cpu(int *a,int *b,int *c)
{
    for(int i=0;i<n;i++)
    {
        c[i] = a[i] +b[i];
    }
}
int main()
{
    int *h_a,*h_b,*h_c;
    int *d_a,*d_b,*d_c;


    clock_t begin,end;
    double time_spent;

    printf("Enter size of vector ");
    scanf("%d",&n);
    
    begin = clock();
    h_a = (int *)malloc(sizeof(int)*n);
    h_b = (int *)malloc(sizeof(int)*n);
    h_c = (int *)malloc(sizeof(int)*n);

    hipMalloc((void **)&d_a,sizeof(int)*n);
    hipMalloc((void **)&d_b,sizeof(int)*n);
    hipMalloc((void **)&d_c,sizeof(int)*n);

    for(int i=0;i<n;i++)
    {
        h_a[i] =rand()%100;
        h_b[i] =rand()%100;
    }


    
    vector_add_cpu(h_a,h_b,h_c);
    end = clock();
    time_spent = (double)(end-begin)/CLOCKS_PER_SEC;
    printf("\nTime spent for Serial Vector Addition %f seconds",time_spent);

    begin = clock();
    hipMemcpy(d_a,h_a,sizeof(int)*n,hipMemcpyHostToDevice);
    hipMemcpy(d_b,h_b,sizeof(int)*n,hipMemcpyHostToDevice);
    
    vector_add_gpu<<<1,n>>>(d_a,d_b,d_c);
    
    hipMemcpy(h_c,d_c,sizeof(int)*n,hipMemcpyDeviceToHost);
    end = clock();

    

    time_spent = (double)(end-begin)/CLOCKS_PER_SEC;

    printf("\nTime spent for Parrellel Vector Addition %f seconds",time_spent);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    free(h_a);
    free(h_b);
    free(h_c);
    return 0;
}